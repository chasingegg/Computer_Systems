
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include <math.h>
#define REP_TIME 100
#define TILE 32
#define SIDE 8
float *initial_matrix(int rows, int cols);
void print_matrix(float *matrix, int rows,int cols);
void check(float *src, float *dst, int rows,int cols);

__global__ void kernel(float *d_src, float *d_dst, int cols, int rows) {

    __shared__ float mat[TILE][TILE+1];

    //block start element
     int bx=blockIdx.x*TILE;  //block start x
     int by=blockIdx.y*TILE;  //block start y
     //thread element
     int i = by+ threadIdx.y;
     int j = bx+ threadIdx.x;

    #pragma unroll
      for(int k=0;k<TILE;k+=SIDE){
        if(i+k<rows&&j<cols)
          mat[threadIdx.y+k][threadIdx.x]=d_src[((i+k)*cols)+j]; 
      }

    __syncthreads();

    int ti=bx+threadIdx.y;
    int tj=by+threadIdx.x;
    #pragma unroll
    for(int k=0;k<TILE;k+=SIDE){
      if((ti+k)<cols&&tj<rows)
        d_dst[(ti+k)*rows+tj]=mat[threadIdx.x][threadIdx.y+k];

    }

}

int main(int argc, char *argv[]) {
    int rows, cols;
    if (argc >= 3) {
        rows = atoi(argv[1]);
        cols = atoi(argv[2]);
    } else {
        rows = 4096;
        cols = 4096;
    }


    //initialization
    float *src, *dst;
    float *d_src, *d_dst;
    src = initial_matrix(rows, cols);
    dst = (float *) malloc(rows * cols * sizeof(float));

    //size_t pitch;
    //cudaMallocPitch(&d_src, &pitch, cols * sizeof(int), rows);
    //Upload Data
    int size = rows * cols * sizeof(float);
    hipMalloc((void **) &d_src, size);
    hipMalloc((void **) &d_dst, size);
    hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);

    //Kernel
    dim3 GridDim(cols/TILE, rows/TILE);
    dim3 BlockDim(TILE, SIDE); //we don't need so TILE*TILE threads in fact

    //count time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(int i=0;i<REP_TIME;++i)
    	kernel <<< GridDim, BlockDim >>> (d_src, d_dst, cols, rows);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);

   printf("Bandwidth:%.4fGB/s\nTotal Time:%.4f(ms)\nIter:%d\nSize:(%d,%d)\n",
	 size*2.0*1000.0/1024/1024/1024/(kernelTime/REP_TIME),
	 kernelTime,
	 REP_TIME,
	 rows,cols);

    //Download Data
    hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost);
    check(src,dst,rows,cols);


  //  print_matrix(src,rows,cols);
  //  printf("\n\n\n");
  //  print_matrix(dst,cols,rows);

    hipFree(d_src);
    hipFree(d_dst);
    free(src);
    free(dst);

    return 0;
}

float *initial_matrix(int rows, int cols) {
    float *pointer = (float *) malloc(rows * cols * sizeof(float));
    for (int i = 0; i < rows * cols; ++i)
        pointer[i] = rand() % 100;
    return pointer;
}

void check(float *src, float *dst, int rows,int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            if (abs(src[i * cols + j] - dst[j * rows + i]) > 0.01) {
                printf("Result dismatch\n");
                return;
            }
        }
    }
    printf("\nResult match!\n");
    return;
}

void print_matrix(float *matrix, int rows,int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%.0f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}
