
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>


float **initial_matrix(int dim);

void transpose(float **src, float **dst, int dim);

void free_matrix(float **pointer, int dim);

void print(float **pointer, int dim);

int main(int argc, char *argv[]) {
    hipEvent_t start, finish;
    float duration;

    float **src, **dst;

    int dim = 4096;
    hipEventCreate(&start);
    hipEventCreate(&finish);
    src = initial_matrix(dim);
    dst = initial_matrix(dim);
    hipEventRecord(start, 0);
    for (int i = 0; i < 100; ++i) {
        transpose(src, dst, dim);
    }
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);
    hipEventElapsedTime(&duration, start, finish);
    
    printf("Time:%.4f(ms)\n",duration/100);
    printf("Bandwidth:%.4f(GB/s)\n", dim*dim*1000.0*2.0*sizeof(float)/1024/1024/1024/(duration/100.0));
   // print(src,dim);
   // print(dst,dim);
    free_matrix(src, dim);
    free_matrix(dst, dim);


    return 0;
}

float **initial_matrix(int dim) {
    float **pointer = new float *[dim];
    for (int i = 0; i < dim; ++i) {
        pointer[i] = new float[dim];

        //fill the matrix
        for (int j = 0; j < dim; ++j) {
            pointer[i][j] = rand() % 100;
        }
    }
    return pointer;
}

void transpose(float **src, float **dst, int dim) {
    for (int i = 0; i < dim; ++i) {
        for (int j = 0; j < dim; ++j)
            dst[i][j] = src[j][i];
    }
}

void free_matrix(float **pointer, int dim) {
    for (int i = 0; i < dim; ++i) {
        free(pointer[i]);
    }
    free(pointer);
}

void print(float **pointer, int dim) {
    for (int i = 0; i < dim; ++i) {
        for (int j = 0; j < dim; ++j) {
            printf("%.1f", pointer[i][j]);
        }
        printf("\n");
    }
}
