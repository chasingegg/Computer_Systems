
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include <math.h>
#define REP_TIME 100
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32
float *initial_matrix(int rows, int cols);
void print_matrix(float *matrix, int rows,int cols);
void check(float *src, float *dst, int rows,int cols);

__global__ void kernel(float *d_src, float *d_dst, int cols, int rows) {
    
    __shared__ float mat[BLOCK_SIZE_X][BLOCK_SIZE_Y+1];

    //block start element
    int bx=blockIdx.x*BLOCK_SIZE_X;  //block start x
    int by=blockIdx.y*BLOCK_SIZE_Y;  //block start y
    //thread element
    int i = by+ threadIdx.y;
    int j = bx+ threadIdx.x;
    //transfered element
    int ti=bx+threadIdx.y;
    int tj=by+threadIdx.x;
   
    //load element to corresponding block
    if(i<rows&&j<cols)
	mat[threadIdx.y][threadIdx.x]=d_src[i*cols+j];
    __syncthreads();
 
    if(tj<rows&&ti<cols)	
	d_dst[ti*rows+tj]=mat[threadIdx.x][threadIdx.y];
}

int main(int argc, char *argv[]) {
    int rows, cols;
    if (argc >= 3) {
        rows = atoi(argv[1]);
        cols = atoi(argv[2]);
    } else {
        rows = 4096;
        cols = 4096;
    }


    //initialization
    float *src, *dst;
    float *d_src, *d_dst;
    src = initial_matrix(rows, cols);
    dst = (float *) malloc(rows * cols * sizeof(float));

    //size_t pitch;
    //cudaMallocPitch(&d_src, &pitch, cols * sizeof(int), rows);
    //Upload Data
    int size = rows * cols * sizeof(float);
    hipMalloc((void **) &d_src, size);
    hipMalloc((void **) &d_dst, size);
    hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);

    //Kernel
    dim3 GridDim(cols/32, rows/32);
    dim3 BlockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y);//notice at most 32*32=1024 threads per block

    //count time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(int i=0;i<REP_TIME;++i)
    	kernel <<< GridDim, BlockDim >>> (d_src, d_dst, cols, rows);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, stop);

    printf("Bandwidth:%.4fGB/s\nTotal Time:%.4f(ms)\nIter:%d\nSize:(%d,%d)\n",
	 size*2.0*1000.0/1024/1024/1024/(kernelTime/REP_TIME),
	 kernelTime,
	 REP_TIME,
	 rows,cols);

    //Download Data
    hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost);
    check(src,dst,rows,cols);
	
    /*
    print_matrix(src,rows,cols);
    printf("\n\n\n");
    print_matrix(dst,cols,rows);
    */
    hipFree(d_src);
    hipFree(d_dst);
    free(src);
    free(dst);

    return 0;
}

float *initial_matrix(int rows, int cols) {
    float *pointer = (float *) malloc(rows * cols * sizeof(float));
    for (int i = 0; i < rows * cols; ++i)
        pointer[i] = rand() % 100;
    return pointer;
}

void check(float *src, float *dst, int rows,int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            if (abs(src[i * cols + j] - dst[j * rows + i]) > 0.01) {
                printf("Result dismatch\n");
                return;
            }
        }
    }
    printf("\nResult match!\n");
    return;
}

void print_matrix(float *matrix, int rows,int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%.1f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

